#include "hip/hip_runtime.h"
#include "Particle.hpp"

__host__ __device__ Particle::Particle()
    : x(0), y(0), z(0), vx(0), vy(0), vz(0) {}

__host__ __device__ Particle::Particle(double x, double y, double z, double vx, double vy, double vz, double mass, double radius)
    : x(x), y(y), z(z), vx(vx), vy(vy), vz(vz), mass(mass), radius(radius) {}

__host__ __device__ void Particle::updatePosition(double timestep)
{
    x += vx * timestep;
    y += vy * timestep;
    z += vz * timestep;
}

__global__ void updateSystemKernel(Particle *particles, int num_particles, double timestep)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_particles)
    {
        particles[i].updatePosition(timestep);
    }
}